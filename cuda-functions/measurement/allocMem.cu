void allocMem() {
  size_t alloc_size = 7000000000;
  int *huge_array;

  if (hipMallocManaged(&huge_array, alloc_size) == hipSuccess)
    if (hipMemset(huge_array, 0, alloc_size) == hipSuccess) {
      hipDeviceSynchronize();
      hipFree(huge_array);
    } else {
      hipFree(huge_array);
    }
  else
}
