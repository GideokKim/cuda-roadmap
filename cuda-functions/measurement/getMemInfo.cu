#include <iostream>
#include <string>

#include "getMemInfo.cuh"

namespace measurement {

int printAllGpuMemInfo() {
  int num_gpus;
  hipGetDeviceCount(&num_gpus);

  size_t free_mem, total_mem;
  for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
    hipSetDevice(gpu_id);
    int id;
    hipGetDevice(&id);
    std::cout << "Activated GPU ID: " << id << std::endl;
    hipError_t result = hipMemGetInfo(&free_mem, &total_mem);
    if (result == hipSuccess) {
      std::cout << "Free memory: " << free_mem << " BYTE" << std::endl;
      std::cout << "Total memory: " << total_mem << " BYTE" << std::endl;
    } else {
      std::string cudaErrorString(hipGetErrorString(result));
      std::cerr << "CUDA Error Code: " << result << std::endl;
      std::cerr << "CUDA Error Message: " << cudaErrorString << std::endl;
      return static_cast<int>(result);
    }
  }
  return 0;
}

// hipError_t getMemInfo(size_t* free_mem, size_t* total_mem) {
//   size_t alloc_size = 40000000000;
//   int* huge_array;

//   if (hipMallocManaged(&huge_array, alloc_size) == hipSuccess)
//     if (hipMemset(huge_array, 0, alloc_size) == hipSuccess) {
//       hipError_t result = hipMemGetInfo(free_mem, total_mem);
//       hipDeviceSynchronize();
//       hipFree(huge_array);
//     } else {
//       hipFree(huge_array);
//     }

//   return result;
// }
}  // namespace measurement
