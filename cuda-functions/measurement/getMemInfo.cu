#include <hip/hip_runtime.h>

#include "getMemInfo.cuh"

int getMemInfo(size_t* free_mem, size_t* total_mem) {
  hipError_t result = hipMemGetInfo(free_mem, total_mem);

  if (result != hipSuccess) {
    return 1;
  }

  return 0;
}
