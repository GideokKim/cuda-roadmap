#include <iostream>
#include <string>

#include "getMemInfo.cuh"

namespace measurement {

void printGpuMemoryInfo() {
  std::vector<GpuMemoryInfo> gpuMemoryInfoList = getGpuMemoryInfo();

  for (size_t i = 0; i < gpuMemoryInfoList.size(); ++i) {
    std::cout << "GPU ID " << i << ": " << std::endl;
    std::cout << "  Total Memory: " << gpuMemoryInfoList[i].totalMemory
              << " BYTE" << std::endl;
    std::cout << "  Free Memory: " << gpuMemoryInfoList[i].freeMemory << " BYTE"
              << std::endl;
  }
}

std::vector<GpuMemoryInfo> getGpuMemoryInfo() {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  std::vector<GpuMemoryInfo> memoryInfoList;

  for (int gpu_id = 0; gpu_id < deviceCount; ++gpu_id) {
    hipSetDevice(gpu_id);
    int id;
    hipGetDevice(&id);
    std::cout << "Activated GPU ID: " << id << std::endl;

    size_t freeMem = 0;
    size_t totalMem = 0;

    hipError_t result = getActivatedGpuMemInfo(&freeMem, &totalMem);
    if (result == hipSuccess) {
      GpuMemoryInfo info{freeMem, totalMem};
      memoryInfoList.push_back(info);
    } else {
      std::string cudaErrorString(hipGetErrorString(result));
      std::cerr << "CUDA Error Code: " << result << std::endl;
      std::cerr << "CUDA Error Message: " << cudaErrorString << std::endl;
    }
  }

  return memoryInfoList;
}

hipError_t getActivatedGpuMemInfo(size_t* free_mem, size_t* total_mem) {
  return hipMemGetInfo(free_mem, total_mem);
}
// hipError_t getMemInfo(size_t* free_mem, size_t* total_mem) {
//   size_t alloc_size = 40000000000;
//   int* huge_array;

//   if (hipMallocManaged(&huge_array, alloc_size) == hipSuccess)
//     if (hipMemset(huge_array, 0, alloc_size) == hipSuccess) {
//       hipError_t result = hipMemGetInfo(free_mem, total_mem);
//       hipDeviceSynchronize();
//       hipFree(huge_array);
//     } else {
//       hipFree(huge_array);
//     }

//   return result;
// }
}  // namespace measurement
