#include <iostream>

#include "asyncMemManager.cuh"

namespace memory {
AsyncMemManager::AsyncMemManager(size_t size)
    : size_(size), d_ptr_(nullptr), stream_(nullptr) {
  // Create a CUDA stream
  hipError_t result = hipStreamCreate(&stream_);
  if (result != hipSuccess) {
    std::cerr << "Failed to create CUDA stream" << result << std::endl;
    std::string cudaErrorString(hipGetErrorString(result));
    std::cerr << "CUDA Error Code: " << result << std::endl;
    std::cerr << "CUDA Error Message: " << cudaErrorString << std::endl;
  } else {
    std::cout << "succeded to create CUDA stream" << std::endl;
  }
}

AsyncMemManager::~AsyncMemManager() {
  if (d_ptr_ != nullptr) {
    hipFreeAsync(d_ptr_, stream_);
  }
  if (stream_ != nullptr) {
    hipStreamDestroy(stream_);
  }
}

void AsyncMemManager::MallocMemoryAsync() {
  if (d_ptr_ == nullptr) {
    hipError_t result = hipMallocAsync(&d_ptr_, size_, stream_);
    if (result != hipSuccess) {
      std::string cudaErrorString(hipGetErrorString(result));
      std::cerr << "CUDA Error Code: " << result << std::endl;
      std::cerr << "CUDA Error Message: " << cudaErrorString << std::endl;
    } else {
      std::cout << "Allocation complete" << std::endl;
    }
  } else {
    std::cout << "Already allocated" << std::endl;
  }
}

void AsyncMemManager::FreeMemoryAsync() {
  if (d_ptr_ != nullptr) {
    hipError_t result = hipFreeAsync(d_ptr_, stream_);
    if (result != hipSuccess) {
      std::string cudaErrorString(hipGetErrorString(result));
      std::cerr << "CUDA Error Code: " << result << std::endl;
      std::cerr << "CUDA Error Message: " << cudaErrorString << std::endl;
    } else {
      std::cout << "Free complete" << std::endl;
      d_ptr_ = nullptr;
    }
  } else {
    std::cout << "Already released" << std::endl;
  }
}

void AsyncMemManager::SyncronizeStream() { hipStreamSynchronize(stream_); }
}  // namespace memory
