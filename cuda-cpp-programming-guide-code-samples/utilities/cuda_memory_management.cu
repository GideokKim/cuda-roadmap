#include <iostream>

#include "cuda_memory_management.cuh"

void allocateDeviceMemory(int **d_data, int size) {
  hipError_t err = hipMalloc(d_data, size * sizeof(int));
  if (err != hipSuccess) {
    std::cerr << "Error allocating device memory: " << hipGetErrorString(err)
              << std::endl;
  }
}

void freeDeviceMemory(int *d_data) { hipFree(d_data); }

void copyHostToDevice(int *h_data, int *d_data, int size) {
  hipError_t err =
      hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "Error copying from host to device: "
              << hipGetErrorString(err) << std::endl;
  }
}

void copyDeviceToHost(int *d_data, int *h_data, int size) {
  hipError_t err =
      hipMemcpy(h_data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr << "Error copying from device to host: "
              << hipGetErrorString(err) << std::endl;
  }
}