#include <iostream>

#include "cuda_memory_management.cuh"

template <typename T>
void allocateDeviceMemory(T **d_data, int size) {
  hipError_t err = hipMalloc(d_data, size * sizeof(T));
  if (err != hipSuccess) {
    std::cerr << "Error allocating device memory: " << hipGetErrorString(err)
              << std::endl;
  }
}

template <typename T>
void freeDeviceMemory(T *d_data) {
  hipFree(d_data);
}

template <typename T>
void copyHostToDevice(T *h_data, T *d_data, int size) {
  hipError_t err =
      hipMemcpy(d_data, h_data, size * sizeof(T), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    std::cerr << "Error copying from host to device: "
              << hipGetErrorString(err) << std::endl;
  }
}

template <typename T>
void copyDeviceToHost(T *d_data, T *h_data, int size) {
  hipError_t err =
      hipMemcpy(h_data, d_data, size * sizeof(T), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    std::cerr << "Error copying from device to host: "
              << hipGetErrorString(err) << std::endl;
  }
}

// 템플릿 인스턴스화
template void allocateDeviceMemory<int>(int **d_data, int size);
template void freeDeviceMemory<int>(int *d_data);
template void copyHostToDevice<int>(int *h_data, int *d_data, int size);
template void copyDeviceToHost<int>(int *d_data, int *h_data, int size);

template void allocateDeviceMemory<float>(float **d_data, int size);
template void freeDeviceMemory<float>(float *d_data);
template void copyHostToDevice<float>(float *h_data, float *d_data, int size);
template void copyDeviceToHost<float>(float *d_data, float *h_data, int size);